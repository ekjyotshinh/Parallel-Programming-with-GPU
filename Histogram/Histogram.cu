#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define NUM_BINS 4096
#define BLOCK_SIZE 512 

#define CUDA_CHECK(ans)                                                  \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
	bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
			file, line);
		if (abort)
			exit(code);
	}
}

__global__ void histogram(unsigned int* input, unsigned int* bins,
	unsigned int num_elements,
	unsigned int num_bins) {
	//@@ the kernel that computes the histogram
	//@@ useing the privitization technique
	__shared__ unsigned int private_histo[NUM_BINS];

	//	if (threadIdx.x < num_bins) private_histo[threadIdx.x] = 0;
	for (int i = 0; i < NUM_BINS / BLOCK_SIZE; i++)
	{
		private_histo[threadIdx.x + blockDim.x * i] = 0;
	}
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	// stride is total number of threads
	//int stride = blockDim.x * gridDim.x;
	if (i < num_elements)
		atomicAdd(&(private_histo[input[i]]), 1);

	__syncthreads();
	for (int j = 0; j < NUM_BINS / BLOCK_SIZE; j++) {
		atomicAdd(&(bins[threadIdx.x + blockDim.x * j]), private_histo[threadIdx.x + blockDim.x * j]);
	}


}

__global__ void saturate(unsigned int* bins, unsigned int num_bins) {
	//@@ the kernel that applies saturtion to counters (i.e., if the bin value is more than 127, make it equal to 127)
	for (int j = 0; j < num_bins / BLOCK_SIZE; j++)
	{
		if (bins[threadIdx.x + blockDim.x * j] > 127)
		{
			bins[threadIdx.x + blockDim.x * j] = 127;
		}
	}
}

int main(int argc, char* argv[]) {
	wbArg_t args;
	int inputLength;
	unsigned int* hostInput;
	unsigned int* hostBins;
	unsigned int* deviceInput;
	unsigned int* deviceBins;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int*)wbImport(wbArg_getInputFile(args, 0),
		&inputLength, "Integer");
	hostBins = (unsigned int*)malloc(NUM_BINS * sizeof(unsigned int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating device memory");
	//@@ Allocate device memory 
	int size = sizeof(float) * inputLength;
	hipMalloc((void**)&deviceInput, size);
	int sizeBin = sizeof(float) * NUM_BINS;
	hipMalloc((void**)&deviceBins, sizeBin);
	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating device memory");

	wbTime_start(GPU, "Copying input host memory to device");
	//@@ Copy input host memory to device
	hipMemcpy(deviceInput, hostInput, size, hipMemcpyHostToDevice);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input host memory to device");

	wbTime_start(GPU, "Clearing the bins on device");
	//@@ zero out the deviceBins using hipMemset() 
	hipMemset(deviceBins, 0, sizeBin);
	wbTime_stop(GPU, "Clearing the bins on device");

	//@@ Initialize the grid and block dimensions 
	dim3 dimBlock(BLOCK_SIZE, 1);
	dim3 dimGrid((inputLength - 1) / BLOCK_SIZE + 1, 1, 1);

	wbLog(TRACE, "Launching kernel");

	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Invoke kernels: first call histogram kernel and then call saturate kernel
	histogram << <dimGrid, dimBlock >> > (deviceInput, deviceBins, inputLength, NUM_BINS);
	saturate << <(1, 1, 1), dimBlock >> > (deviceBins, NUM_BINS);

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output device memory to host");
	//@@ Copy output device memory to host
	hipMemcpy(hostBins, deviceBins, sizeBin, hipMemcpyDeviceToHost);


	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output device memory to host");

	wbTime_start(GPU, "Freeing device memory");
	//@@ Free the device memory here
	hipFree(deviceInput);
	hipFree(deviceBins);

	wbTime_stop(GPU, "Freeing device memory");

	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);
	return 0;
}