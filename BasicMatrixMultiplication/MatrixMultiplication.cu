#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <wb.h>

#define TILE_WIDTH 16 //do not change this value

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBColumns) {
    // Calculate the row and column index for the C matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute the value for C if the indices are within bounds
    if (row < numARows && col < numBColumns) {
        float value = 0.0f;
        for (int k = 0; k < numAColumns; k++) {
            value += A[row * numAColumns + k] * B[k * numBColumns + col];
        }
        C[row * numBColumns + col] = value;
    }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  hostC = NULL;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;      // Rows of C = Rows of A
  numCColumns = numBColumns;    // Columns of C = Columns of B
  //@@ Allocate the hostC matrix
  hostC = (float*)malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here

  wbCheck(hipMalloc((void**)&deviceA, numARows * numAColumns * sizeof(float)));
  wbCheck(hipMalloc((void**)&deviceB, numBRows * numBColumns * sizeof(float)));
  wbCheck(hipMalloc((void**)&deviceC, numCRows * numCColumns * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here

  wbCheck(hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(16, 16); // Using 16x16 thread blocks
  dim3 gridDim((numCColumns + blockDim.x - 1) / blockDim.x, (numCRows + blockDim.y - 1) / blockDim.y);


  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply <<<gridDim, blockDim >>> (deviceA, deviceB, deviceC, numARows, numAColumns, numBColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  wbCheck(hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  wbCheck(hipFree(deviceA));
  wbCheck(hipFree(deviceB));
  wbCheck(hipFree(deviceC));

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
